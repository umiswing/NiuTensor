/*
Use the Welford algorithm to calculate the mean
and variance along a dimension of the tensor
>> input - the input tensor
>> mean - the tensor store the mean
>> var - the tensor store the variance
>> dim - the dimension where the Welford is performed on
*/
#include "Welford.h"
#include "../../XDevice.h"
#include "../../XTensor.h"
#include "Welford.cuh"
#include "../../XName.h"
namespace nts {

void Welford(const XTensor &input, XTensor &mean, XTensor &var, int dim) {

  CheckNTErrors(dim >= 0 && dim < input.order, "Illegal dimension to reduce!");

  if (!mean.isInit || !XTensor::IsReduceShaped(&input, &mean, dim)) {
    int order = input.order - 1;
    int *dimSize = new int[order];
    for (int i = 0; i < order; i++) {
      if (i < dim)
        dimSize[i] = input.dimSize[i];
      else if (i >= dim)
        dimSize[i] = input.dimSize[i + 1];
    }

    float dr = (!input.isSparse) ? 1.0F : input.denseRatio;
    InitTensorV2(&mean, order, dimSize, input.dataType, dr, input.devID,
                 input.mem);

    /* destroy variables */
    delete[] dimSize;
  }
  if (!var.isInit || !XTensor::IsReduceShaped(&input, &var, dim)) {
    int order = input.order - 1;
    int *dimSize = new int[order];
    for (int i = 0; i < order; i++) {
      if (i < dim)
        dimSize[i] = input.dimSize[i];
      else if (i >= dim)
        dimSize[i] = input.dimSize[i + 1];
    }

    float dr = (!input.isSparse) ? 1.0F : input.denseRatio;
    InitTensorV2(&var, order, dimSize, input.dataType, dr, input.devID,
                 input.mem);

    /* destroy variables */
    delete[] dimSize;
  }
  mean.SetTMPFlag();
  var.SetTMPFlag();
  int stride = 1;
  int strideNum = input.dimSize[dim];
  int blockSize = 1;
  int blockNum = 1;
  for (int i = 0; i < input.order; i++) {
    if (i < dim)
      blockNum *= input.dimSize[i];
    else if (i > dim)
      stride *= input.dimSize[i];
  }
  blockSize = stride * strideNum;
  // view the input tensor as a 2-order tensor with shape (blockNum,
  // strideNum*stride)
  constexpr int thread_group_width = kWarpSize;
  const int cols_per_thread =
      (strideNum * stride + thread_group_width - 1) / thread_group_width;
  constexpr int rows_per_access = 1;
  constexpr int pack_size = 1;

  int devID = input.devID;
  int devIDBackup;
  ProtectCudaDev(devID, devIDBackup);
  #if 0
  hipStream_t Stream;
  hipStreamCreate(&Stream);
  #endif
  DispatchLayerNormWarpImpl<float *, int, float>(
      0, (float *)input.data, 1, blockNum, stride * strideNum, 1,
      (float *)mean.data, (float *)var.data);
  #if 0
  hipStreamDestroy(Stream);
  #endif
#if 0
  _Welford<pack_size, 1, thread_group_width, rows_per_access, true>
      <<<1, 1>>>((float *)input.data, (float *)mean.data, (float *)var.data,
                 blockNum, stride * strideNum);
#endif
  BacktoCudaDev(devID, devIDBackup);

  if (input.enableGrad) {
    /* tensor connections */
    XLink::MakeLink(&input, NULL, &mean, REDUCE_REDUCEMEAN);
    XLink::AddParamToHeadInt(&mean, dim);

    XLink::MakeLink(&input, &mean, &var, REDUCE_REDUCEVARIANCE);
    XLink::AddParamToHeadInt(&var, dim);
  }
}
} // namespace nts
